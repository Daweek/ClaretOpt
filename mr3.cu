#include "hip/hip_runtime.h"
/*Archivo mr3.cu que contiene el codigo para CUDA.
    Renderizacion por OpenGL-CUDA interoperability
    Nucleo del codigo para calcular la fuerza entre particulas
    Creado por: Martinez Noriega Edgar Josafat
*/
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// ***** CUDA includes
#include <cutil.h>

#define GL_ON
#define KER
#define NMAX      8192
#define NTHRE       64
#define ATYPE        8
#define ATYPE2    (ATYPE * ATYPE)
#define ThreadsPB 64
//////For NaCl Optminized if_kernel
#define NTHREOPT      256
#define NDIVBIT      4
#define NDIV      (1<<NDIVBIT)
#define NTHREOPT2    (NTHREOPT/NDIV)

typedef struct {
  float r[3];
  int atype;
} VG_XVEC;

typedef struct {
  float pol;
  float sigm;
  float ipotro;
  float pc;
  float pd;
  float zz;
} VG_MATRIX;


/////////GLOBAL Variables/////////////////////////////////////////
int   *d_atypemat;
VG_XVEC *d_x=NULL;
int mem_flg=0;
int mem_flg2=0;
int mem_sp=5;
int mem_cpu=0;
int flg1=0,flg2=0,flg3=0;

//////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////
////////FORCE CALCULATION WITH GPU/////////////////////////////////////
//////////////////////////////////////////////////////////////////////

__global__
void update_coor_kernel(int n3, float *vl,VG_XVEC *cd,float *xs,
                        float *fc,float *side){
#ifdef KER
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n3){
            vl[tid]   =  (vl[tid]*(1-(*xs))+fc[tid])/(1+(*xs));
            cd[tid/3].r[tid % 3]   +=   vl[tid];
			if (cd[tid/3].r[tid % 3] < 0 || cd[tid/3].r[tid % 3] > side[tid % 3]) vl[tid] *= -1;
    }
#endif
}

//////////////////NaCl Optmized
///////////////////////////////

__constant__ VG_MATRIX c_matrix[4]={
[0].pol=1.250000,[0].sigm=2.340000,[0].ipotro=3.154574,[0].pc=0.072868,[0].pd=0.034699,[0].zz=1.000000,
[1].pol=1.000000,[1].sigm=2.755000,[1].ipotro=3.154574,[1].pc=0.485784,[1].pd=0.602893,[1].zz=-1.000000,
[2].pol=1.000000,[2].sigm=2.755000,[2].ipotro=3.154574,[2].pc=0.485784,[2].pd=0.602893,[2].zz=-1.000000,
[3].pol=0.750000,[3].sigm=3.170000,[3].ipotro=3.154574,[3].pc=5.031334,[3].pd=10.106042,[3].zz=1.000000,
};

__device__ __inline__
void inter_if(float xj[3], float xi[3], float fi[3], int t, float xmax,
		float xmax1) {
#ifdef KER
	int k;
	float dn2, r, inr, inr2, inr4, inr8, d3, dr[3];
	float pb = (float) (0.338e-19 / (14.39 * 1.60219e-19)), dphir;

	dn2 = 0.0f;
	for (k = 0; k < 3; k++) {
		dr[k] = xi[k] - xj[k];
		dr[k] -= rintf(dr[k] * xmax1) * xmax;
		dn2 += dr[k] * dr[k];
	}
	r = sqrtf(dn2);
#if 1
	inr = 1.0f / r;
#elif 0
	if(dn2 != 0.0f) inr = 1.0f / r;
	else inr = 0.0f;
#elif 0
	if(dn2 == 0.0f) inr = 0.0f;
	else inr = 1.0f / r;
#else
	inr = 1.0f / r;
	if(dn2 == 0.0f) inr = 0.0f;
#endif
	inr2 = inr * inr;
	inr4 = inr2 * inr2;
	inr8 = inr4 * inr4;
	d3 = pb * c_matrix[t].pol
			* expf((c_matrix[t].sigm - r) * c_matrix[t].ipotro);
	dphir =
			(d3 * c_matrix[t].ipotro * inr - 6.0f * c_matrix[t].pc * inr8
					- 8.0f * c_matrix[t].pd * inr8 * inr2
					+ inr2 * inr * c_matrix[t].zz);
#if 1
	if (dn2 == 0.0f)
		dphir = 0.0f;
#endif
	for (k = 0; k < 3; k++)
		fi[k] += dphir * dr[k];
#endif
}

__global__
void nacl_kernel_if2(VG_XVEC *x, int n, int nat, float xmax, float *fvec) {
#ifdef KER
	int tid = threadIdx.x;
	int jdiv = tid / NTHREOPT2;
	int i = blockIdx.x * NTHREOPT2 + (tid & (NTHREOPT2 - 1));
	int j, k;
	float xmax1 = 1.0f / xmax;
	int atypei;
	float xi[3];
	__shared__ VG_XVEC s_xj[NTHREOPT];
	__shared__ float s_fi[NTHREOPT][3];

	for (k = 0; k < 3; k++)
		s_fi[tid][k] = 0.0f;
	for (k = 0; k < 3; k++)
		xi[k] = x[i].r[k];
	atypei = x[i].atype * nat;
	int na;
	na = n / NTHREOPT;
	na = na * NTHREOPT;
	for (j = 0; j < na; j += NTHREOPT) {
		__syncthreads();
		s_xj[tid] = x[j + tid];
		__syncthreads();
#pragma unroll 16
		for (int js = jdiv; js < NTHREOPT; js += NDIV)
			inter_if(s_xj[js].r, xi, s_fi[tid], atypei + s_xj[js].atype, xmax,
					xmax1);
	}
	for (j = na + jdiv; j < n; j += NDIV) {
		inter_if(x[j].r, xi, s_fi[tid], atypei + x[j].atype, xmax, xmax1);
	}
#if NTHREOPT>=512 && NTHREOPT2<=256
	__syncthreads();
	if(tid<256) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+256][k];
#endif
#if NTHREOPT>=256 && NTHREOPT2<=128
	__syncthreads();
	if (tid < 128)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 128][k];
#endif
#if NTHREOPT>=128 && NTHREOPT2<=64
	__syncthreads();
	if (tid < 64)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 64][k];
#endif
#if NTHREOPT>=64 && NTHREOPT2<=32
	__syncthreads();
	if (tid < 32)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 32][k];
#endif
#if NTHREOPT2<=16
	if (tid < 16)
		for (k = 0; k < 3; k++)
			s_fi[tid][k] += s_fi[tid + 16][k];
#endif
#if NTHREOPT2<=8
	if(tid<8) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+8][k];
#endif
#if NTHREOPT2<=4
	if(tid<4) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+4][k];
#endif
#if NTHREOPT2<=2
	if(tid<2) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+2][k];
#endif
#if NTHREOPT2<=1
	if(tid<1) for(k=0;k<3;k++) s_fi[tid][k]+=s_fi[tid+1][k];
#endif
	if (jdiv == 0)
		for (k = 0; k < 3; k++)
			fvec[i * 3 + k] = s_fi[tid][k];
#endif
}

 __global__
void rem_offset_kernell (int n3, float *force){
#ifdef KER
	int tid = threadIdx.x + blockIdx.x *blockDim.x;

	float center [3];
	center[0]=0.0;
	center[1]=0.0;
	center[2]=0.0;

	if(tid < n3/3) {
			center[0]=force[tid*3];
			center[1]=force[tid*3+1];
			center[2]=force[tid*3+2];
	 }

	center[0]/=n3/3;
	center[1]/=n3/3;
	center[2]/=n3/3;

    if (tid < n3/3){
		 force[tid*3]-= center[0];
		 force[tid*3+1]-= center[1];
		 force[tid*3+2]-=  center[2];
	}
#endif
}


__global__
void velforce_kernel(int n3, float *fc, float *a_mass, float *vl,
                     VG_XVEC *atype, int *atype_mat, float hsq,float *ekin1){
#ifdef KER
	__shared__ float cache [ThreadsPB];
    int indx = threadIdx.x;
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;

	float tmp = 0;

    if (tid < n3 ){
		fc[tid] *= hsq/a_mass[atype_mat[atype[tid/3].atype]];

	}

	if(tid < n3/3){
        tmp  = (vl[tid*3]*vl[tid*3 ]    +
                vl[tid*3+1]*vl[tid*3+1]	+
                vl[tid*3+2]*vl[tid*3+2])* a_mass[atype_mat[atype[tid].atype]];

    }

	cache [indx] = tmp;
    __syncthreads();

    int i = blockDim.x/2;

	while (i != 0){

        if (indx < i) cache[indx] += cache [indx + i];
        __syncthreads();
        i /= 2;
    }

	if (indx == 0) ekin1[blockIdx.x] = cache [0];
#endif
}


__global__
void serie_kernel (	float *ekin,float *mtemp,float *mpres,float *xs,float tscale,
                    float nden, float vir,int s_num,int w_num,float rtemp,
					float lq,float hsq,float *ekin1a, int limi){


#ifdef KER
		float aux = 0;
		float aux1 = *xs;

		for(int p=0;p<limi;p++)aux += ekin1a[p];
		*ekin = aux;
		*ekin /= hsq;
        *mtemp = tscale * (*ekin);
        *mpres  = nden / 3.f * ((*ekin) - (vir)) / (s_num + w_num);
        aux1 += (*mtemp - rtemp) /  lq * hsq *.5f;
		*xs = aux1;
#endif
}



extern "C"
void mdlop(int n3,int grape_flg,double phi [3],double *phir,double *iphi, double *vir,int s_num3,
			timeval time_v,double *md_time0,double *md_time,int *m_clock,int md_step,double *mtemp,
			double tscale,double *mpres,double nden,int s_num,int w_num,double rtemp,double lq,
			double x[], int n, int atype[], int nat,
			double pol[], double sigm[], double ipotro[],
		 	double pc[], double pd[],double zz[],
		 	int tblno, double xmax, int periodicflag,
		 	double force[],
			double hsq,double a_mass [], int atype_mat [], double *ekin,double *vl,
			double *xs,double side []){

//////////////VARIABLES FROM HE BEGINING/////////////////
  int md_loop;
/////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////
  int i,j;
  float *d_force=NULL;
  float xmaxf;
  //VG_MATRIX *matrix=NULL;
  VG_XVEC   *vec=NULL;
  if((periodicflag & 1)==0) xmax*=2.0;
  xmaxf=xmax;
  float *forcef=NULL;
  int n_bak=0;

/////////////////////////////////////////////////////////

	int  blocksPGrid = (n3 + ThreadsPB - 1)/(ThreadsPB);
	dim3 THREADS(NTHRE);
	dim3 BLOCKS((n3 + ThreadsPB - 1)/(ThreadsPB));
	dim3 threads(NTHREOPT);
	dim3 grid((n * NDIV + NTHREOPT - 1) / NTHREOPT);


	float   *d_side;
	float   fxs = *xs;
	float   fside[3],*ffc;
	float   *vla;

	float   *d_amass,*d_vl;
	int     p = 0;
	float   hsqf = hsq;
	float   *fvl,fa_mass[4];

	ffc = (float*)malloc(n3*sizeof(float));
	fvl = (float*)malloc(n3*sizeof(float));

	float *d_ekin1,*ekin1a,ekinaux;

	float ftscale = tscale,fnden = nden,frtemp = rtemp,flq = lq,fvir = 0;
	float fmtemp = *mtemp,fmpres = *mpres;
	float *d_ekin,*d_xs,*d_mtemp,*d_mpres;

	for (p=0;p<4;p++) fa_mass[p] = (float) a_mass[p];
	for (p=0;p<3;p++) fside[p] = (float) side[p];
	for (p=0;p<n3;p++){
		fvl     [p] =  (float) *(vl +p);
		ffc     [p] =  (float) *(force +p);
	}
/////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////


  if(sizeof(double)*n*3<sizeof(VG_MATRIX)*nat*nat){
    fprintf(stderr,"** ethreadIdx.xrror : n*3<nat*nat **\n");
    exit(1);
  }
  if(nat>ATYPE){
    fprintf(stderr,"** error : nat is too large **\n");
    exit(1);
  }

  if(n!=n_bak){
    int nalloc;
    int nalloc_bak=0;
    if(n>NMAX) nalloc=n;
    else       nalloc=NMAX;
    if(nalloc!=nalloc_bak){
  		vec   =(VG_XVEC*)  malloc((nalloc+NTHREOPT2)*sizeof(VG_XVEC));
  		CUDA_SAFE_CALL(hipMalloc((void**)&d_x,sizeof(VG_XVEC)* (nalloc + NTHREOPT2)));
  		CUDA_SAFE_CALL(hipMalloc((void**)&d_force,sizeof(float)*(nalloc + NTHREOPT2)*3));
  		free(forcef);
  		if((forcef=(float *)malloc(sizeof(float)*nalloc*3))==NULL){
    	  fprintf(stderr,"** error : can't malloc forcef **\n");
    	  exit(1);
  		}

  	memset(forcef,0,sizeof(float)*nalloc*3);
    nalloc_bak=nalloc;
    }

	n_bak=n;
  }


	for (i = 0; i < (n + NTHREOPT2 - 1) / NTHREOPT2 * NTHREOPT2; i++) {
		if (i < n) {
			for (j = 0; j < 3; j++) {
				vec[i].r[j] = x[i * 3 + j];
			}
			vec[i].atype = atype[i];
		} else {
			for (j = 0; j < 3; j++) {
				vec[i].r[j] = 0.0f;
			}
			vec[i].atype = 0;
		}
	}

///////////////////////////////////////////////////////////////////
 // ensure force has enough size for temporary array
  	if(sizeof(double)*n*3<sizeof(float)*nat*nat){
    	fprintf(stderr,"** error : n*3<nat*nat **\n");
    	exit(1);
  	}
//////////////////////////
////////// allocate global memory and copy from host to GPU

    vla 	= (float*)malloc(n3*sizeof(float));
	ekin1a 	= (float*)malloc(blocksPGrid*sizeof(float));

#if 1
	CUDA_SAFE_CALL(hipMalloc((void**)&d_side,3*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_amass,4*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_vl,n3*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_atypemat,20*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin,sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_xs,sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_mtemp,sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_mpres,sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_ekin1,blocksPGrid*sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpy(d_x,vec,sizeof(VG_XVEC)*((n + NTHREOPT2 - 1) / NTHREOPT2 * NTHREOPT2),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_side,fside,sizeof(float)*3,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_mtemp,&fmtemp,sizeof(float),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_mpres,&fmpres,sizeof(float),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_xs,&fxs,sizeof(float),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_vl,fvl,sizeof(float)*n3,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_amass,fa_mass,sizeof(float)*4,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_atypemat,atype_mat,sizeof(int)*20,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_force,ffc,sizeof(float)*n*3,hipMemcpyHostToDevice));

///////Md_loop///////////////////////////////////////////////
	for(md_loop = 0; md_loop < md_step; md_loop++){
    	*m_clock+=1;

    	gettimeofday(&time_v,NULL);
    	*md_time0 = (time_v.tv_sec + time_v.tv_usec / 1000000.0);

    	update_coor_kernel<<<BLOCKS,THREADS>>>(n3,d_vl,d_x,d_xs,d_force,d_side);
		nacl_kernel_if2<<<grid, threads>>>(d_x, n, nat, xmaxf, d_force);

		rem_offset_kernell<<<BLOCKS,THREADS>>>(n3,d_force);
		velforce_kernel<<<BLOCKS,THREADS>>>(n3,d_force,d_amass,d_vl,d_x,d_atypemat,hsqf,d_ekin1);
		serie_kernel<<<1,1>>>(d_ekin,d_mtemp,d_mpres,d_xs,ftscale,fnden,fvir,s_num,w_num,frtemp,flq,hsqf,d_ekin1,blocksPGrid);
		hipDeviceSynchronize();

		gettimeofday(&time_v,NULL);
		*md_time = (time_v.tv_sec + time_v.tv_usec / 1000000.0);

	}

/////////////////Copy back to the CPU
	CUDA_SAFE_CALL(hipMemcpy(forcef,d_force,sizeof(float)*n*3,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(vla,d_vl,n3*sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(&fxs,d_xs,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(&ekinaux,d_ekin,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(&fmtemp,d_mtemp,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(&fmpres,d_mpres,sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(vec,d_x,n*sizeof(VG_XVEC),hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(d_vl));
    CUDA_SAFE_CALL(hipFree(d_amass));
    CUDA_SAFE_CALL(hipFree(d_atypemat));
	CUDA_SAFE_CALL(hipFree(d_xs));
	CUDA_SAFE_CALL(hipFree(d_ekin));
	CUDA_SAFE_CALL(hipFree(d_mtemp));
	CUDA_SAFE_CALL(hipFree(d_mpres));
	CUDA_SAFE_CALL(hipFree(d_ekin1));
    CUDA_SAFE_CALL(hipFree(d_x));
    CUDA_SAFE_CALL(hipFree(d_force));
#endif

	for(i=0;i<n;i++) for(j=0;j<3;j++) force[i*3+j]=(double) forcef[i*3+j];
    for(p=0;p<n3;p++) {
        *(vl+p) = (double) vla[p];
    }

	for(i=0;i<n;i++){
    	for(j=0;j<3;j++){
      	*(x+i*3+j)= (double)vec[i].r[j];
    	}
  	}

	*xs 	= (double) fxs;
	*ekin 	= (double) ekinaux;
	*mtemp 	= (double) fmtemp;
	*mpres 	= (double) fmpres;
/////////////////////////////////////////////////////////
// free allocated global memory


	//free(matrix);
	free(vec);
    free(forcef);
	free(vla);
	free(ekin1a);

}

